#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <stdio.h>

__global__ void addVectorsKernel(const float *a, const float *b, float *c,
                                 int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void cuda_vector_add_impl(const float *a, const float *b, float *c, int n) {
    // Basic device checking
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n", deviceCount);

    // Print input values for debugging
    printf("First values: a[0]=%f, b[0]=%f\n", a[0], b[0]);

    // Allocate device memory
    float *d_a = NULL, *d_b = NULL, *d_c = NULL;
    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_b, n * sizeof(float));
    hipMalloc((void **)&d_c, n * sizeof(float));

    // Check allocations
    if (d_a == NULL || d_b == NULL || d_c == NULL) {
        printf("Memory allocation failed\n");
        return;
    }

    // Copy to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    printf("Launching kernel with %d blocks, %d threads\n", numBlocks,
           blockSize);
    addVectorsKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Synchronize and check errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    } else {
        printf("Kernel execution successful\n");
    }

    // Copy back to host
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    printf("Result: c[0]=%f\n", c[0]);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
