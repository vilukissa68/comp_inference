#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <stdio.h>

#define CHECK_CUDA(call)                                                       \
    {                                                                          \
        hipError_t err = call;                                                \
        if (err != hipSuccess) {                                              \
            fprintf(stderr, "CUDA error in %s (%s:%d): %s\n", #call, __FILE__, \
                    __LINE__, hipGetErrorString(err));                        \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

__global__ void addVectorsKernel(const float *a, const float *b, float *c,
                                 int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

void cuda_vector_add_impl(const float *a, const float *b, float *c, int n) {
    // Basic device checking
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Device count: %d\n", deviceCount);

    // Print input values for debugging
    printf("First values: a[0]=%f, b[0]=%f\n", a[0], b[0]);

    // Allocate device memory
    float *d_a = NULL, *d_b = NULL, *d_c = NULL;
    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_b, n * sizeof(float));
    hipMalloc((void **)&d_c, n * sizeof(float));

    // Check allocations
    if (d_a == NULL || d_b == NULL || d_c == NULL) {
        printf("Memory allocation failed\n");
        return;
    }

    // Copy to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    printf("Launching kernel with %d blocks, %d threads\n", numBlocks,
           blockSize);
    addVectorsKernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Synchronize and check errors
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    } else {
        printf("Kernel execution successful\n");
    }

    // Copy back to host
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    printf("Result: c[0]=%f\n", c[0]);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

__global__ void simulate_decompression_kernel(const input_type *input,
                                              output_type *output,
                                              uint64_t input_size,
                                              uint64_t output_size) {
    uint64_t idx =
        uint64_t(blockIdx.x) * uint64_t(blockDim.x) + uint64_t(threadIdx.x);
    if (idx < output_size) {
        if (idx < input_size) {
            output[idx] = static_cast<output_type>(input[idx]) + 1;
        } else {
            output[idx] = 2;
        }
    }
}

void cuda_simulate_decompression(const input_type *input, output_type *output,
                                 uint64_t input_size, uint64_t output_size) {
    int deviceId = 0; // Change this if you want to use a different GPU
    CHECK_CUDA(hipSetDevice(deviceId));

    // Basic device checking
    int deviceCount;
    CHECK_CUDA(hipGetDeviceCount(&deviceCount));
    printf("Device count: %d\n", deviceCount);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);
    printf("maxGridDimX = %d, maxGridDimY = %d, maxGridDimZ = %d\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    // Allocate device memory
    input_type *d_input = nullptr;
    output_type *d_output = nullptr;

    float memcpy_time_ms = 0.0f;
    float decomp_time_ms = 0.0f;
    hipEvent_t memcpy_start, memcpy_stop, decomp_start, decomp_stop;
    hipEventCreate(&memcpy_start);
    hipEventCreate(&memcpy_stop);
    hipEventCreate(&decomp_start);
    hipEventCreate(&decomp_stop);

    CHECK_CUDA(hipMalloc((void **)&d_input, input_size * sizeof(input_type)));
    CHECK_CUDA(
        hipMalloc((void **)&d_output, output_size * sizeof(output_type)));

    hipEventRecord(memcpy_start);
    CHECK_CUDA(hipMemcpy(d_input, input, input_size * sizeof(input_type),
                          hipMemcpyHostToDevice));
    hipEventRecord(memcpy_stop);
    hipEventSynchronize(memcpy_stop);
    hipEventElapsedTime(&memcpy_time_ms, memcpy_start, memcpy_stop);

    // Kernel launch configuration
    uint64_t threads_per_block = 256;
    uint64_t num_blocks =
        (output_size + threads_per_block - 1ULL) / threads_per_block;
    printf("Launching kernel with %llu blocks, %llu threads\n", num_blocks,
           threads_per_block);

    hipEventRecord(decomp_start);
    simulate_decompression_kernel<<<num_blocks, threads_per_block>>>(
        d_input, d_output, input_size, output_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Launch error: %s\n", hipGetErrorString(err));
    }

    CHECK_CUDA(hipDeviceSynchronize());
    hipEventRecord(decomp_stop);
    hipEventSynchronize(decomp_stop);
    hipEventElapsedTime(&decomp_time_ms, decomp_start, decomp_stop);

    printf("Decompression kernel time: %f ms\n", decomp_time_ms);
    printf("Memory copy time: %f ms\n", memcpy_time_ms);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    } else {
        printf("Decompression kernel execution successful\n");
    }

    CHECK_CUDA(hipMemcpy(output, d_output, output_size * sizeof(output_type),
                          hipMemcpyDeviceToHost));

    hipFree(d_input);
    hipFree(d_output);
}
